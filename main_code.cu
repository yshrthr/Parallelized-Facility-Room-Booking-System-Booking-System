	#include <iostream>
	#include <stdio.h>
	#include <hip/hip_runtime.h>

	#define max_N 100000
	#define max_P 30
	#define BLOCKSIZE 1024
  	#define VAL 100000
  	#define LARGE_VAL 1000000

	using namespace std;


//storing VAL in lock array
	__global__ void get_lock(int* lock, int start,int end)
	{
    int tid = threadIdx.x; 
    int bid = blockIdx.x * blockDim.x + start;    
    		
    if((tid + bid)  < end)
		lock[tid+bid] = VAL;
	}

  
//Kernel to fill slots array 
__global__ void fill_slots(int second, int first, int end, int *d_slots, int* d_capacity)
{
    int tid = threadIdx.x; 
    int bid = blockIdx.x * blockDim.x + second; 
    int id = tid+bid; 

    if(id<end)
      d_slots[id] = d_capacity[first+(id-second)/24]; 
}


//Main kernel to find total successful requests and successfull requests in each centre
__global__ void count_success_req(int *d_succ_reqs, int *d_succ, int begin, int R,int first, int second, int *d_offset,int c,int *d_room_offset,bool check, int *d_lock, int volatile *d_slots, int *d_req_cen, int *d_req_fac, int *d_req_start, int *d_req_slots) 
{
    int id = blockIdx.x *blockDim.x + threadIdx.x;
    
    bool flag = false, execute = true;
    int stop;
    __shared__ bool done;
    id += begin;
    done = true;

    int slot_offset = d_offset[d_req_cen[id]]+d_req_fac[id]*(c+1)*24;
    int room_offset = d_room_offset[d_req_cen[id]]+d_req_fac[id];

    __syncthreads();

    int x = d_req_start[id]-first;
    int y = d_req_start[id]+d_req_slots[id] - second;
    

    while(check) 
    {  
        if (execute != false) 
          atomicMin(&d_lock[room_offset], id);

        else check = true;
        
        done = true;

        __syncthreads();

        if (execute != false) { 
            if (d_lock[room_offset] == id) {
                int begin = slot_offset + x;
                int end = slot_offset + y;
                int temp = slot_offset/24;

                for (int i = begin; i <= end; i++){
                    atomicSub((unsigned int*)&d_slots[i], 1);
                    if (d_slots[i] < c) {
                        flag = true;
                        stop = i;
                        break;
                    }
                }
                

                if (flag == true) {
                      int end2 = stop;

                      while(begin<=end2) {
                        atomicAdd((unsigned int*)&d_slots[begin], 1);
                        begin++;
                      }
                    temp++;
                    flag = false;
                }
                
                else {
                    atomicInc((unsigned int*) &d_succ[0], 100001);
                    atomicInc((unsigned int*) &d_succ_reqs[d_req_cen[id]], 100001);
                }
                
                d_lock[room_offset] = LARGE_VAL;
                execute = false;
            } 

            else if (execute != false) 
             done = false;

            else check = true; 

        }
        __syncthreads();

        if(done)
        {
          check = false;
        } 
    } 
}

//Boiler plate code begins

int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	  int R;
	  fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ((R)*sizeof (int));  // Request ids
    req_cen = (int *) malloc ((R)*sizeof (int));  // Requested computer centre
    req_fac = (int *) malloc ((R)*sizeof (int));  // Requested facility
    req_start = (int *) malloc((R)*sizeof (int));  // Start slot of every request
    req_slots = (int *) malloc((R)*sizeof (int));   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		
//Boiler plate code ends-------------------------------------------
//Kernel calling code begins

    int v1=0, v2=0;

    //Arrays on host 
  	int *offset = (int*)malloc(N * sizeof (int));
	int *room_offset=(int*)malloc(N * sizeof (int));

    //Declaring device arrays
    int *d_offset, *d_total_succ, *d_succ_reqs, *d_room_offset,*d_lock,*d_req_start, *d_req_cen, *d_req_fac, *d_req_slots;

		// Allocating memory on gpu
    hipMalloc(&d_lock, k1 * sizeof(int));
    hipMalloc(&d_total_succ, sizeof(int));
	hipMalloc(&d_req_cen,R * sizeof(int));
	hipMalloc(&d_req_fac,R * sizeof(int));
	hipMalloc(&d_req_start,R * sizeof(int));
	hipMalloc(&d_req_slots,R * sizeof(int));
	hipMalloc(&d_succ_reqs,N * sizeof(int));
    hipMalloc(&d_room_offset, N * sizeof(int));
	hipMalloc(&d_offset, N*sizeof(int));

    //Device array to keep track of capicity
    int *d_capacity;
    hipMalloc(&d_capacity, max_P*N * sizeof(int));
	hipMemcpy(d_capacity, capacity, k1 * sizeof(int), hipMemcpyHostToDevice);

    //Device array to keep track of empty slots
    int *d_slots;
    hipMalloc(&d_slots,max_P*N *24* sizeof(int));


    for (int i = 0; i < N; i++) {

      int num_facility = facility[i];
      int end = v2 + 24 * num_facility;

      room_offset[i] = v1;
      offset[i] = v2;

      int nb = ceil(float(end - v2+1)/BLOCKSIZE);

      fill_slots<<<nb,BLOCKSIZE>>>(v2, v1, end, d_slots, d_capacity);
      hipDeviceSynchronize();

      v1 += num_facility;
      v2 = end;
    }


	hipMemcpy(d_offset, offset, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_room_offset, room_offset, k1*24 * sizeof(int), hipMemcpyHostToDevice);

    //function for getting lock
	int num_blocks = ceil((float(k1)/BLOCKSIZE));
    int end = k1;
	get_lock<<<num_blocks,BLOCKSIZE>>>(d_lock,0, end);	

      //This host array stores total successful requests
    int *h_total_succ;
    h_total_succ = (int*)malloc(sizeof (int));

			//Host to device copy
	hipMemcpy(d_req_fac, req_fac, R * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_req_cen, req_cen, R*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_req_slots,req_slots, R * sizeof(int),hipMemcpyHostToDevice);
    hipMemset(d_total_succ, 0, sizeof(int));
	hipMemcpy(d_req_start,req_start, R * sizeof(int),hipMemcpyHostToDevice);		
	hipMemcpy(d_succ_reqs, succ_reqs, N* sizeof(int), hipMemcpyHostToDevice);
			
    bool check = true; 
    int nblocks = R/BLOCKSIZE, remaining = R % BLOCKSIZE;
    int total_blocks = 0, blocks = 1;

	//Calling kernel for processing requests in parallel in a batch of 1024
	while(total_blocks<nblocks) {	
    int begin = total_blocks*BLOCKSIZE;
	count_success_req<<<blocks,BLOCKSIZE>>>(d_succ_reqs,d_total_succ, begin, R, 1, 2, d_offset,0, d_room_offset, check,d_lock, d_slots, d_req_cen, d_req_fac, d_req_start, d_req_slots );
	hipDeviceSynchronize();
	total_blocks++;
	}

    //Calling kernel if any remaining requests
    if(remaining !=0){
    	int begin = BLOCKSIZE*nblocks;
		count_success_req<<<blocks,remaining>>>(d_succ_reqs,d_total_succ, begin, R,1,2, d_offset,0, d_room_offset, check,d_lock, d_slots, d_req_cen, d_req_fac, d_req_start, d_req_slots);
    }
      
    //get back the result
	hipMemcpy(succ_reqs, d_succ_reqs, N* sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_total_succ, d_total_succ, sizeof(int), hipMemcpyDeviceToHost);
      
    success = h_total_succ[0];
    fail = R-success;
   //printf("%d %d\n", success, fail);
  //kernel calling and computation ends-------------------
    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}
